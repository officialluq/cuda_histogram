#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>

const int numBins = 256;

__global__ void calculateHistogram(const unsigned char* inputImage, int* histogramR, int* histogramG, int* histogramB, int imageSize) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < imageSize) {
        atomicAdd(&histogramR[inputImage[3 * tid]], 1);
        atomicAdd(&histogramG[inputImage[3 * tid + 1]], 1);
        atomicAdd(&histogramB[inputImage[3 * tid + 2]], 1);
    }
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <image_file_path>" << std::endl;
        return -1;
    }

    const char* imagePath = argv[1];

    std::ifstream file(imagePath, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open the image file." << std::endl;
        return -1;
    }

    file.seekg(0, std::ios::end);
    int imageSize = file.tellg();
    file.seekg(0, std::ios::beg);

    unsigned char* h_image = new unsigned char[imageSize];
    file.read(reinterpret_cast<char*>(h_image), imageSize);
    file.close();

    unsigned char* d_image;
    hipMalloc(&d_image, imageSize * sizeof(unsigned char));
    hipMemcpy(d_image, h_image, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    int* d_histogramR;
    int* d_histogramG;
    int* d_histogramB;
    hipMalloc(&d_histogramR, numBins * sizeof(int));
    hipMalloc(&d_histogramG, numBins * sizeof(int));
    hipMalloc(&d_histogramB, numBins * sizeof(int));
    hipMemset(d_histogramR, 0, numBins * sizeof(int));
    hipMemset(d_histogramG, 0, numBins * sizeof(int));
    hipMemset(d_histogramB, 0, numBins * sizeof(int));

    int blockSize = 32;
    int gridSize = (imageSize/3 + blockSize - 1) / blockSize;
    std::cout << "Size: " << gridSize << std::endl;

    // Run kernel
    calculateHistogram<<<gridSize, blockSize>>>(d_image, d_histogramR, d_histogramG, d_histogramB, imageSize);

    int h_histogramR[numBins];
    int h_histogramG[numBins];
    int h_histogramB[numBins];
    hipMemcpy(h_histogramR, d_histogramR, numBins * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_histogramG, d_histogramG, numBins * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_histogramB, d_histogramB, numBins * sizeof(int), hipMemcpyDeviceToHost);

    std::ofstream csvFileR("histogram_red.csv");
    std::ofstream csvFileG("histogram_green.csv");
    std::ofstream csvFileB("histogram_blue.csv");
    if (csvFileR.is_open() && csvFileG.is_open() && csvFileB.is_open()) {
        csvFileR << "Bin,Count" << std::endl;
        csvFileG << "Bin,Count" << std::endl;
        csvFileB << "Bin,Count" << std::endl;

        for (int i = 0; i < numBins; ++i) {
            csvFileR << i << "," << h_histogramR[i] << std::endl;
            csvFileG << i << "," << h_histogramG[i] << std::endl;
            csvFileB << i << "," << h_histogramB[i] << std::endl;
        }

        csvFileR.close();
        csvFileG.close();
        csvFileB.close();

        std::cout << "Histogram data saved to histogram_red.csv, histogram_green.csv, and histogram_blue.csv" << std::endl;
    } else {
        std::cerr << "Error: Could not open the CSV files for writing." << std::endl;
    }

    // Free alloc mem
    delete[] h_image;
    hipFree(d_image);
   
}